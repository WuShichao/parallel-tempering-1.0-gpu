/*
 * pmcmc_fsv.cu
 *
 *  Created on: 27-Jan-2010
 *      Author: alee
 */

#include <stdio.h>
//#include <cutil.h>
#include "rng.h"
#include "gauss.h"
#include "output.h"
#include "kalman.h"
#include "matrix.h"
#include "fsv.h"
#include "smc_fsv.h"
#include "smc_lg.h"
#include "smc_usv.h"
#include "smc_mvlg.h"
#include "scan.h"
#include "usv.h"

void test_smc_fsv_forget(int N, int Dx, int Dy, int T, float* ys_real, float* scale_step,
        float* cov_step, float* h_args_l, int nb, int nt) {
//    unsigned int hTimer;
//    double time;
//    cutCreateTimer(&hTimer);

    float* d_xs;
    float* x_init;
    float* d_ys_real;

    float* xs = (float*) malloc(N * Dx * sizeof(float));
    hipMalloc((void**) &d_xs, N * Dx * sizeof(float));
    hipMalloc((void**) &x_init, N * Dx * sizeof(float));
    hipMalloc((void**) &d_ys_real, T * Dy * sizeof(float));
    hipMemcpy(d_ys_real, ys_real, T * Dy * sizeof(float), hipMemcpyHostToDevice);

    float* ws = (float*) malloc(N * sizeof(float));
    float* d_ws;
    hipMalloc((void**) &d_ws, N * sizeof(float));

    //  populate_randn_d(x_init, N * D);

    float* hx_init = (float*) malloc(N * Dx * sizeof(float));
    matrix_zero(hx_init, N, Dx);
    hipMemcpy(x_init, hx_init, N * Dx * sizeof(float), hipMemcpyHostToDevice);
    free(hx_init);

    float ll_forget_fsv;

//    cutResetTimer(hTimer);
//    cutStartTimer(hTimer);

    smc_forget_fsv(x_init, d_xs, d_ws, d_ys_real, N, Dx, Dy, T, h_args_l, scale_step, cov_step,
            &ll_forget_fsv, nb, nt);

//    cutStopTimer(hTimer);
//    time = cutGetTimerValue(hTimer);
//    printf("Time = %f, ", time);

    printf("ll_fsv_forget = %f\n", ll_forget_fsv);

    free(ws);
    free(xs);
    hipFree(d_xs);
    hipFree(x_init);
    hipFree(d_ys_real);
}

void test_fsv(int M, int N, int T, int nb, int nt) {
    const int Dx = 3;
    const int Dy = 5;

    float theta[Dx + Dx * Dx + Dy * Dy]

    float scale_step[Dx * Dx];
    matrix_identity(scale_step, Dx);
    matrix_times(scale_step, scale_step, 0.9f, Dx, Dx);

    float cov_step[Dx * Dx] = { 0.5f, 0.2f, 0.1f, 0.2f, 0.5f, 0.2f, 0.1f, 0.2f, 0.5f };

    float Psi[Dy * Dy];
    matrix_identity(Psi, Dy);
    matrix_times(Psi, Psi, 0.5f, Dy, Dy);

    float B[Dy * Dx] = { 1.0f, 0.0f, 0.0f, 0.5f, 1.0f, 0.0f, 0.5f, 0.5f, 1.0f, 0.2f, 0.6f, 0.3f,
            0.8f, 0.7f, 0.5f };

    float* xs_real = (float*) malloc(T * Dx * sizeof(float));
    float* ys_real = (float*) malloc(T * Dy * sizeof(float));

    kill_rng();
    seed_rng(16384, 32, 128);

    generate_data_fsv(xs_real, ys_real, Dx, Dy, T, scale_step, cov_step, Psi, B);

    printf("%f\n", xs_real[T - 1]);

    to_file(xs_real, T * Dx, "fsv_xs_real.txt");
    to_file(ys_real, T * Dy, "fsv_ys_real.txt");

    float h_args_l[Dy * Dx + Dx * Dy + Dy * Dy];
    matrix_transpose(B, h_args_l + Dy * Dx, Dy, Dx);
    for (int i = 0; i < Dy * Dx; i++) {
        h_args_l[i] = B[i];
    }
    for (int i = 0; i < Dy * Dy; i++) {
        h_args_l[2 * Dy * Dx + i] = Psi[i];
    }

    kill_rng();
    seed_rng(8192, 32, 128);

    test_smc_fsv_forget(N, Dx, Dy, T, ys_real, scale_step, cov_step, h_args_l, nb, nt);

    free(xs_real);
    free(ys_real);
}

int main(int argc, char **argv) {

    //    int N = 8192;
    //    int N = 16384;
    //    int N = 32768;
    //    int N = 65536;
    //    int N = 262144;

    //    int N = 8192;
    //   int N = 16384;
    //    int N = 32768;
            int N = 4096;

//    int N = 131072;

    int nb = 32;
    int nt = 128;

    int T = 200;

    seed_rng(8192, 32, 128);

    scan_init(N);

    test_fsv(1, N, T, nb, nt);

    kill_rng();
    scan_destroy();
}
