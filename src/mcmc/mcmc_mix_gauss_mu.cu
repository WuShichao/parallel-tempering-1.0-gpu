#include "hip/hip_runtime.h"
/*
 * mcmc_mix_gauss_mu.cu
 *
 *  Created on: 26-Feb-2009
 *      Author: alee
 */

#include "mcmc_mix_gauss_mu.h"

#include "func.h"
#include "mix_gauss_uniform.ch"

#define TARGET mgu_mu_pdf
#define LOG_TARGET log_mgu_mu_pdf
#define TYPE mgumu_mv
#define NUM_AP 8198

#include "mcmc_kernel_mv.cu"

void FUNC(metropolis_rwpop, TYPE)(int N, int D, float* d_array_init, float sigma,
		float* h_args_p, float* d_temps, float* d_array_out, int log, int nb,
		int nt) {
	switch (D) {
	case 1:
		FUNC(metropolis_rwpop, TYPE)<1,1>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
		break;
	case 2:
		FUNC(metropolis_rwpop, TYPE)<2,1>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
		break;
	case 3:
		FUNC(metropolis_rwpop, TYPE)<3,1>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
		break;
	case 4:
		FUNC(metropolis_rwpop, TYPE)<4,1>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);

	default:
		break;
		}
		
}

void FUNC(metropolis_rw, TYPE)(int N, int D, float* d_array_init, float sigma,
		float* d_array_out, float* h_args_p, int log, int nb, int nt) {
	switch (D) {
	case 1: FUNC(metropolis_rw, TYPE)<1>(N, d_array_init, sigma, d_array_out, h_args_p, log, nb, nt);
			break;
	case 2: FUNC(metropolis_rw, TYPE)<2>(N, d_array_init, sigma, d_array_out, h_args_p, log, nb, nt);
			break;
	case 3: FUNC(metropolis_rw, TYPE)<3>(N, d_array_init, sigma, d_array_out, h_args_p, log, nb, nt);
			break;
	case 4: FUNC(metropolis_rw, TYPE)<4>(N, d_array_init, sigma, d_array_out, h_args_p, log, nb, nt);
			break;
	default:
		break;
	}
}

void FUNC(metropolis_rwpop_marginal, TYPE)(int N, int D, float* d_array_init,
		float sigma, float* h_args_p, float* d_temps, float* d_array_out,
		int log, int nb, int nt, int red) {
	switch (D) {
	case 1: FUNC(metropolis_rwpop_marginal, TYPE)<1,128>(N, d_array_init, sigma, h_args_p, d_temps,
			d_array_out, log, nb, nt);
			break;
	case 2: FUNC(metropolis_rwpop_marginal, TYPE)<2,128>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
	case 3: FUNC(metropolis_rwpop_marginal, TYPE)<3,128>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
	case 4: 

		switch (red) {
			case 32768:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,32768>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 16384:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,16384>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 8192:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,8192>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 4096:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,4096>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 2048:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,2048>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 1024:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,1024>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 512:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,512>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 256:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,256>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 128:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,128>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 64:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,64>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 32:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,32>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 16:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,16>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 8:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,8>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 4:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,4>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 2:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,2>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			case 1:
				FUNC(metropolis_rwpop_marginal, TYPE)<4,1>(N, d_array_init, sigma, h_args_p, d_temps,
				d_array_out, log, nb, nt);
				break;
			default:
				break;
			}	
	default:
		break;
	}
}
